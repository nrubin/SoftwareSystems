#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
  #include "utils.h"
 
    __global__
  void gaussian_blur(const unsigned char* const inputChannel,
                     unsigned char* const outputChannel,
                     int numRows, int numCols,
                     const float* const filter, const int filterWidth)
  {
 
    const int pixel_col = blockIdx.x;
    const int pixel_row = threadIdx.x;
 
    int offset_pixel_row, offset_pixel_col;
 
    // iterate through the kernel
    float acc = 0.f;
    for (int col=0; col<filterWidth; col++) {
      for (int row=0; row<filterWidth; row++) {
        offset_pixel_row = pixel_row - filterWidth/2 + row;
        offset_pixel_col = pixel_col - filterWidth/2 + col;
        // clamp to the image boundary
        if (offset_pixel_row < 0) {
          offset_pixel_row = 0;
        } else if (offset_pixel_row > numRows - 1) {
          offset_pixel_row = numRows - 1;
        }   
        if (offset_pixel_col < 0) {
          offset_pixel_col = 0;
        } else if (offset_pixel_col > numCols - 1) {
          offset_pixel_col = numCols - 1;
        }
        acc += filter[filterWidth * row + col] * inputChannel[offset_pixel_row * numCols + offset_pixel_col];
      }
    }
    // assign the 1D output channel the accumulated val
    outputChannel[pixel_row * numCols + pixel_col] = (unsigned char)((int)acc);
  }
 
  //This kernel takes in an image represented as a uchar4 and splits
  //it into three images consisting of only one color channel each
  __global__
  void separateChannels(const uchar4* const inputImageRGBA,
                        int numRows,
                        int numCols,
                        unsigned char* const redChannel,
                        unsigned char* const greenChannel,
                        unsigned char* const blueChannel)
  {
    const int pixel_col = blockIdx.x;
    const int pixel_row = threadIdx.x;
    const int i = pixel_row * numCols + pixel_col;
    redChannel[i] = inputImageRGBA[i].x;
    greenChannel[i] = inputImageRGBA[i].y;
    blueChannel[i] = inputImageRGBA[i].z;
  }
 
  //This kernel takes in three color channels and recombines them
  //into one image.  The alpha channel is set to 255 to represent
  //that this image has no transparency.
  __global__
  void recombineChannels(const unsigned char* const redChannel,
                         const unsigned char* const greenChannel,
                         const unsigned char* const blueChannel,
                         uchar4* const outputImageRGBA,
                         int numRows,
                         int numCols)
  {
    const int pixel_col = blockIdx.x;
    const int pixel_row = threadIdx.x;
    const int thread_1D_pos = pixel_row * numCols + pixel_col;
 
    unsigned char red   = redChannel[thread_1D_pos];
    unsigned char green = greenChannel[thread_1D_pos];
    unsigned char blue  = blueChannel[thread_1D_pos];
 
    //Alpha should be 255 for no transparency
    uchar4 outputPixel = make_uchar4(red, green, blue, 255);
 
    outputImageRGBA[thread_1D_pos] = outputPixel;
  }
 
  unsigned char *d_red, *d_green, *d_blue;
  float         *d_filter;
 
  void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                  const float* const h_filter, const size_t filterWidth)
  {
 
    //allocate memory for the three different channels
    checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
    checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
    checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));
 
    // Allocate memory for the filter on the GPU
    checkCudaErrors(hipMalloc(&d_filter, sizeof(float) * filterWidth * filterWidth));
 
    // Copy the filter on the host (h_filter) to the memory you just allocated
    checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));
 
  }
 
  void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                          uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                          unsigned char *d_redBlurred,
                          unsigned char *d_greenBlurred,
                          unsigned char *d_blueBlurred,
                          const int filterWidth)
  {
    const dim3 blockSize = numRows; //n rows, means n threads per col
    const dim3 gridSize = numCols;  //m cols, means m blocks with n rows (threads) each
 
    separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA,
                                              numRows,
                                              numCols,
                                              d_red,
                                              d_green,
                                              d_blue);
 
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
 
    // perform blurring on each channel
    gaussian_blur<<<gridSize, blockSize>>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
    gaussian_blur<<<gridSize, blockSize>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
    gaussian_blur<<<gridSize, blockSize>>>(d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);
 
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
 
    recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                               d_greenBlurred,
                                               d_blueBlurred,
                                               d_outputImageRGBA,
                                               numRows,
                                               numCols);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  }
 
 
  // free the red, green, and blue channels we allocated
  void cleanup() {
    checkCudaErrors(hipFree(d_red));
    checkCudaErrors(hipFree(d_green));
    checkCudaErrors(hipFree(d_blue));
  }